#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <vector>
#define GRAPH_LAUNCH_ITERATIONS  300

using std::cout;
using std::endl;

#define MASK_LENGTH 7

__constant__ int mask[MASK_LENGTH];

void initialize_vector(int* v, int n) {
    for (int i = 0; i < n; i++) {
        v[i] = rand() % 100;
    }
}

__global__ void convolution_1d(int* array, int* result, int n) {

    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    int r = MASK_LENGTH / 2;

    int start = tid - r;

    int temp = 0;

    for (int j = 0; j < MASK_LENGTH; j++) {
        if (((start + j) >= 0) && (start + j < n)) {
            temp += array[start + j] * mask[j];
        }
    }

    result[tid] = temp;
}

void verify_result(int* array, int* mask, int* result, int n) {
    int radius = MASK_LENGTH / 2;
    int temp;
    int start;
    for (int i = 0; i < n; i++) {
        start = i - radius;
        temp = 0;
        for (int j = 0; j < MASK_LENGTH; j++) {
            if ((start + j >= 0) && (start + j < n)) {
                temp += array[start + j] * mask[j];
            }
        }
        if (temp != result[i]) {
            cout << "NOT SUCCESSFUL" << endl;
            exit(0);
        }
    }
}

void cudaGraphAPIconvolution(int* h_array, int* h_mask, int* h_result, int* d_array, int* d_result, int n, int THREADS, int GRID, size_t bytes_n, size_t bytes_m) {

    initialize_vector(h_array, n);
    hipStream_t streamForGraph;
    hipGraph_t graph;
    std::vector<hipGraphNode_t> nodeDependencies;
    hipGraphNode_t memcpyNode, kernelNode, memsetNode;
    double result_h = 0.0;

    hipStreamCreateWithFlags(&streamForGraph, hipStreamNonBlocking);

    hipKernelNodeParams kernelNodeParams = { 0 };
    hipMemcpy3DParms memcpyParams = { 0 };
    hipMemsetParams memsetParams = { 0 };

    //Adding memcpyParams node
    memcpyParams.srcArray = NULL;
    memcpyParams.srcPos = make_hipPos(0, 0, 0);
    memcpyParams.srcPtr = make_hipPitchedPtr(h_array, sizeof(int) * n, n, 1);
    memcpyParams.dstArray = NULL;
    memcpyParams.dstPos = make_hipPos(0, 0, 0);
    memcpyParams.dstPtr = make_hipPitchedPtr(d_array, sizeof(int) * n, n, 1);
    memcpyParams.extent = make_hipExtent(sizeof(int) * n, 1, 1);
    memcpyParams.kind = hipMemcpyHostToDevice;

    //Adding memsetParams node
    memsetParams.dst = (void*)d_result;
    memsetParams.value = 0;
    memsetParams.pitch = 0;
    memsetParams.elementSize = sizeof(int);
    memsetParams.width = n;
    memsetParams.height = 1;

    hipGraphCreate(&graph, 0);
    hipGraphAddMemcpyNode(&memcpyNode, graph, NULL, 0, &memcpyParams);
    hipGraphAddMemsetNode(&memsetNode, graph, NULL, 0, &memsetParams);
    nodeDependencies.push_back(memsetNode);
    nodeDependencies.push_back(memcpyNode);

    //Adding Kernal node
    void* kernelArgs[3] = { (void*)&d_array, (void*)&d_result, &n };
    kernelNodeParams.func = (void*)convolution_1d;
    kernelNodeParams.gridDim = dim3(GRID, 1, 1);
    kernelNodeParams.blockDim = dim3(THREADS, 1, 1);
    kernelNodeParams.sharedMemBytes = 0;
    kernelNodeParams.kernelParams = (void**)kernelArgs;
    kernelNodeParams.extra = NULL;

    hipGraphAddKernelNode(&kernelNode, graph, nodeDependencies.data(), nodeDependencies.size(), &kernelNodeParams);
    nodeDependencies.clear();
    nodeDependencies.push_back(kernelNode);

    //Adding memcpyParams node
    memset(&memcpyParams, 0, sizeof(memcpyParams));
    memcpyParams.srcArray = NULL;
    memcpyParams.srcPos = make_hipPos(0, 0, 0);
    memcpyParams.srcPtr = make_hipPitchedPtr(d_result, sizeof(int) * n, n, 1);
    memcpyParams.dstArray = NULL;
    memcpyParams.dstPos = make_hipPos(0, 0, 0);
    memcpyParams.dstPtr = make_hipPitchedPtr(h_result, sizeof(int) * n, n, 1);
    memcpyParams.extent = make_hipExtent(sizeof(int) * n, 1, 1);
    memcpyParams.kind = hipMemcpyDeviceToHost;

    hipGraphAddMemcpyNode(&memcpyNode, graph, nodeDependencies.data(), nodeDependencies.size(), &memcpyParams);
    nodeDependencies.clear();
    nodeDependencies.push_back(memcpyNode);

    hipGraphNode_t* nodes = NULL;
    size_t numNodes = 0;
    hipGraphGetNodes(graph, nodes, &numNodes);
    cout << "\nNum of nodes in the graph created manually = " << numNodes << endl;

    hipGraphExec_t graphExec;
    hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0);

    hipGraph_t clonedGraph;
    hipGraphExec_t clonedGraphExec;
    hipGraphClone(&clonedGraph, graph);
    hipGraphInstantiate(&clonedGraphExec, clonedGraph, NULL, NULL, 0);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    for (int i = 0; i < GRAPH_LAUNCH_ITERATIONS; i++)
    {
        hipGraphLaunch(graphExec, streamForGraph);
        hipStreamSynchronize(streamForGraph);
        verify_result(h_array, h_mask, h_result, n);
    }
    
    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    cout << "Verifying Cloned Graph ..." << endl;
    for (int i = 0; i < GRAPH_LAUNCH_ITERATIONS; i++)
    {
        hipGraphLaunch(clonedGraphExec, streamForGraph);
        hipStreamSynchronize(streamForGraph);
        verify_result(h_array, h_mask, h_result, n);
    }
    cout << "Done! Verifyied successfully" << endl;

    cout << "\nTime taken by using CUDA GRAPH in ms : " << milliseconds / GRAPH_LAUNCH_ITERATIONS << endl;
    
    hipGraphExecDestroy(graphExec);
    hipGraphExecDestroy(clonedGraphExec);
    hipGraphDestroy(graph);
    hipGraphDestroy(clonedGraph);
    hipStreamDestroy(streamForGraph);
    hipEventDestroy(start);
    hipEventDestroy(stop);

}

void NormalConvolution(int* h_array, int* h_mask, int* h_result, int* d_array, int* d_result, const int n, int THREADS, int GRID, size_t bytes_n) {

    hipMemcpy(d_array, h_array, bytes_n, hipMemcpyHostToDevice);

    convolution_1d << <GRID, THREADS >> > (d_array, d_result, n);

    hipMemcpy(h_result, d_result, bytes_n, hipMemcpyDeviceToHost);

    verify_result(h_array, h_mask, h_result, n);
}

int main() {

    int n = 1 << 20;

    int bytes_n = n * sizeof(int);
    size_t bytes_m = MASK_LENGTH * sizeof(int);

    int* h_array = new int[n];
    int* h_mask = new int[MASK_LENGTH];
    int* h_result = new int[n];

    int* d_array, * d_result;
    hipMalloc(&d_array, bytes_n);
    hipMalloc(&d_result, bytes_n);
    
    initialize_vector(h_array, n);
    initialize_vector(h_mask, MASK_LENGTH);
    hipMemcpyToSymbol(HIP_SYMBOL(mask), h_mask, bytes_m);

    int THREADS = 256;

    int GRID = (n + THREADS - 1) / THREADS;

    cout << "Normal Convolution" << endl;
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    for (int i = 0; i < GRAPH_LAUNCH_ITERATIONS; i++) {
        NormalConvolution(h_array, h_mask, h_result, d_array, d_result, n, THREADS, GRID, bytes_n);
    }
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    cout << "\nTime taken without CUDA GRAPH in ms : " << milliseconds / GRAPH_LAUNCH_ITERATIONS << endl;
    
    cout << "----------------------------------------------------" << endl;
    
    cout << "Convolution using CUDA GRAPHS (Graph API)" << endl;
    
    cudaGraphAPIconvolution(h_array, h_mask, h_result, d_array, d_result, n, THREADS, GRID, bytes_n, bytes_m);
    
    cout << "----------------------------------------------------\n" << endl;
    cout << "\nThe time is the average time of all the kernel launchs. The total kernel launches are " << GRAPH_LAUNCH_ITERATIONS << endl;
    
    
    delete[] h_array;
    delete[] h_result;
    delete[] h_mask;
    hipFree(d_result);
    hipFree(d_array);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return 0;
}