#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <hip/hip_runtime_api.h>
#define GRAPH_LAUNCH_ITERATIONS  300

using std::cout;
using std::endl;

#define MASK_LENGTH 7

__constant__ int mask[MASK_LENGTH];

void initialize_vector(int* v, int n) {
    for (int i = 0; i < n; i++) {
        v[i] = rand() % 100;
    }
}

__global__ void convolution_1d(int* array, int* result, int n) {

    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    int r = MASK_LENGTH / 2;

    int start = tid - r;

    int temp = 0;

    for (int j = 0; j < MASK_LENGTH; j++) {
        if (((start + j) >= 0) && (start + j < n)) {
            temp += array[start + j] * mask[j];
        }
    }

    result[tid] = temp;
}

void verify_result(int* array, int* mask, int* result, int n) {
    int radius = MASK_LENGTH / 2;
    int temp;
    int start;
    for (int i = 0; i < n; i++) {
        start = i - radius;
        temp = 0;
        for (int j = 0; j < MASK_LENGTH; j++) {
            if ((start + j >= 0) && (start + j < n)) {
                temp += array[start + j] * mask[j];
            }
        }
        if (temp != result[i]) {
            cout << "NOT SUCCESSFUL" << endl;
            exit(0);
        }
    }
}

void StreamCaptureConvolution(int* h_array, int* h_mask, int* h_result, int* d_array, int* d_result, const int n, int THREADS, int GRID, size_t bytes_n, size_t bytes_m) {

    hipStream_t stream1, stream2, streamForGraph;
    hipEvent_t forkStreamEvent, memcpyEvent;
    hipGraph_t graph;
    int result = 0;

    checkCudaErrors(hipStreamCreateWithFlags(&stream1, hipStreamNonBlocking));
    checkCudaErrors(hipStreamCreateWithFlags(&stream2, hipStreamNonBlocking));
    checkCudaErrors(hipStreamCreateWithFlags(&streamForGraph, hipStreamNonBlocking));

    checkCudaErrors(hipEventCreate(&forkStreamEvent));
    checkCudaErrors(hipEventCreate(&memcpyEvent));

    checkCudaErrors(hipStreamBeginCapture(stream1, hipStreamCaptureModeGlobal));
    checkCudaErrors(hipEventRecord(forkStreamEvent, stream1));
    checkCudaErrors(hipStreamWaitEvent(stream2, forkStreamEvent));

    checkCudaErrors(hipMemcpyAsync(d_array, h_array, bytes_n, hipMemcpyDefault, stream2));
    checkCudaErrors(hipEventRecord(memcpyEvent, stream2));
    checkCudaErrors(hipStreamWaitEvent(stream1, memcpyEvent));

    convolution_1d << <GRID, THREADS, 0, stream1 >> > (d_array, d_result, n);

    checkCudaErrors(hipMemcpyAsync(h_result, d_result, bytes_n, hipMemcpyDefault, stream1));

    checkCudaErrors(hipStreamEndCapture(stream1, &graph));

    hipGraphNode_t* nodes = NULL;
    size_t numNodes = 0;
    checkCudaErrors(hipGraphGetNodes(graph, nodes, &numNodes));
    cout << "Num of nodes in the graph created using stream capture API = " << numNodes << endl;

    hipGraphExec_t graphExec;
    checkCudaErrors(hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0));

    hipGraph_t clonedGraph;
    hipGraphExec_t clonedGraphExec;
    checkCudaErrors(hipGraphClone(&clonedGraph, graph));
    checkCudaErrors(hipGraphInstantiate(&clonedGraphExec, clonedGraph, NULL, NULL, 0));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    for (int i = 0; i < GRAPH_LAUNCH_ITERATIONS; i++)
    {
        checkCudaErrors(hipGraphLaunch(graphExec, streamForGraph));
        checkCudaErrors(hipStreamSynchronize(streamForGraph));
        verify_result(h_array, h_mask, h_result, n);
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    cout << "\nVerifying Cloned Graph ... " << endl;
    for (int i = 0; i < GRAPH_LAUNCH_ITERATIONS; i++)
    {
        checkCudaErrors(hipGraphLaunch(clonedGraphExec, streamForGraph));
        checkCudaErrors(hipStreamSynchronize(streamForGraph));
        verify_result(h_array, h_mask, h_result, n);
    }
    cout << "Done! Verifyied successfully" << endl;

    cout << "\nTime taken by using CUDA GRAPH in ms : " << milliseconds / GRAPH_LAUNCH_ITERATIONS << endl;

    checkCudaErrors(hipGraphExecDestroy(graphExec));
    checkCudaErrors(hipGraphExecDestroy(clonedGraphExec));
    checkCudaErrors(hipGraphDestroy(graph));
    checkCudaErrors(hipGraphDestroy(clonedGraph));
    checkCudaErrors(hipStreamDestroy(stream1));
    checkCudaErrors(hipStreamDestroy(stream2));
    checkCudaErrors(hipStreamDestroy(streamForGraph));
    checkCudaErrors(hipEventDestroy(forkStreamEvent));
    checkCudaErrors(hipEventDestroy(memcpyEvent));
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));

}

void NormalConvolution(int* h_array, int* h_mask, int* h_result, int* d_array, int* d_result, const int n, int THREADS, int GRID, size_t bytes_n) {


    hipMemcpy(d_array, h_array, bytes_n, hipMemcpyHostToDevice);

    convolution_1d << <GRID, THREADS >> > (d_array, d_result, n);

    hipMemcpy(h_result, d_result, bytes_n, hipMemcpyDeviceToHost);

    verify_result(h_array, h_mask, h_result, n);

}

int main() {

    int n = 1 << 20;

    int bytes_n = n * sizeof(int);
    size_t bytes_m = MASK_LENGTH * sizeof(int);

    int* h_array = new int[n];
    int* h_mask = new int[MASK_LENGTH];
    int* h_result = new int[n];

    int* d_array, * d_result;
    checkCudaErrors(hipMalloc(&d_array, bytes_n));
    checkCudaErrors(hipMalloc(&d_result, bytes_n));

    initialize_vector(h_array, n);
    initialize_vector(h_mask, MASK_LENGTH);
    hipMemcpyToSymbol(HIP_SYMBOL(mask), h_mask, bytes_m);

    int THREADS = 256;

    int GRID = (n + THREADS - 1) / THREADS;

    cout << "Normal Convolution" << endl;

    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    checkCudaErrors(hipEventRecord(start));

    for (int i = 0; i < GRAPH_LAUNCH_ITERATIONS; i++) {
        NormalConvolution(h_array, h_mask, h_result, d_array, d_result, n, THREADS, GRID, bytes_n);
        checkCudaErrors(hipStreamSynchronize(0));
    }
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipEventRecord(stop));
    checkCudaErrors(hipEventSynchronize(stop));
    float milliseconds = 0;
    checkCudaErrors(hipEventElapsedTime(&milliseconds, start, stop));
    cout << "\nTime taken without CUDA GRAPH in ms : " << milliseconds / GRAPH_LAUNCH_ITERATIONS << endl;

    cout << "----------------------------------------------------" << endl;

    cout << "Convolution using CUDA GRAPHS (Stream Capture)" << endl;

    StreamCaptureConvolution(h_array, h_mask, h_result, d_array, d_result, n, THREADS, GRID, bytes_n, bytes_m);

    cout << "----------------------------------------------------\n" << endl;
    cout << "\nThe time is the average time of all the kernel launchs. The total kernel launches are " << GRAPH_LAUNCH_ITERATIONS << endl;

    delete[] h_array;
    delete[] h_result;
    delete[] h_mask;
    checkCudaErrors(hipFree(d_result));
    checkCudaErrors(hipFree(d_array));
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));

    return 0;
}
