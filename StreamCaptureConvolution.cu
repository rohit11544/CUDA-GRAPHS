#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#define GRAPH_LAUNCH_ITERATIONS  300

using std::cout;
using std::endl;

#define MASK_LENGTH 7

__constant__ int mask[MASK_LENGTH];

void initialize_vector(int* v, int n) {
    for (int i = 0; i < n; i++) {
        v[i] = rand() % 100;
    }
}

__global__ void convolution_1d(int* array, int* result, int n) {

    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    int r = MASK_LENGTH / 2;

    int start = tid - r;

    int temp = 0;

    for (int j = 0; j < MASK_LENGTH; j++) {
        if (((start + j) >= 0) && (start + j < n)) {
            temp += array[start + j] * mask[j];
        }
    }

    result[tid] = temp;
}

void verify_result(int* array, int* mask, int* result, int n) {
    int radius = MASK_LENGTH / 2;
    int temp;
    int start;
    for (int i = 0; i < n; i++) {
        start = i - radius;
        temp = 0;
        for (int j = 0; j < MASK_LENGTH; j++) {
            if ((start + j >= 0) && (start + j < n)) {
                temp += array[start + j] * mask[j];
            }
        }
        if (temp != result[i]) {
            cout << "NOT SUCCESSFUL" << endl;
            exit(0);
        }
    }
}

void StreamCaptureConvolution(int* h_array, int* h_mask, int* h_result, int* d_array, int* d_result, const int n, int THREADS, int GRID, size_t bytes_n, size_t bytes_m) {

    hipStream_t stream1, stream2, streamForGraph;
    hipEvent_t forkStreamEvent, memcpyEvent;
    hipGraph_t graph;
    int result = 0;

    hipStreamCreateWithFlags(&stream1, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&stream2, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&streamForGraph, hipStreamNonBlocking);

    hipEventCreate(&forkStreamEvent);
    hipEventCreate(&memcpyEvent);

    hipStreamBeginCapture(stream1, hipStreamCaptureModeGlobal);
    hipEventRecord(forkStreamEvent, stream1);
    hipStreamWaitEvent(stream2, forkStreamEvent);

    hipMemcpyAsync(d_array, h_array, bytes_n, hipMemcpyDefault, stream2);
    hipEventRecord(memcpyEvent, stream2);
    hipStreamWaitEvent(stream1, memcpyEvent);

    convolution_1d << <GRID, THREADS, 0, stream1 >> > (d_array, d_result, n);

    hipMemcpyAsync(h_result, d_result, bytes_n, hipMemcpyDefault, stream1);

    hipStreamEndCapture(stream1, &graph);

    hipGraphNode_t* nodes = NULL;
    size_t numNodes = 0;
    hipGraphGetNodes(graph, nodes, &numNodes);
    cout << "Num of nodes in the graph created using stream capture API = " << numNodes << endl;

    hipGraphExec_t graphExec;
    hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0);

    hipGraph_t clonedGraph;
    hipGraphExec_t clonedGraphExec;
    hipGraphClone(&clonedGraph, graph);
    hipGraphInstantiate(&clonedGraphExec, clonedGraph, NULL, NULL, 0);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    for (int i = 0; i < GRAPH_LAUNCH_ITERATIONS; i++)
    {
        hipGraphLaunch(graphExec, streamForGraph);
        hipStreamSynchronize(streamForGraph);
        verify_result(h_array, h_mask, h_result, n);
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    cout << "\nVerifying Cloned Graph ... " << endl;
    for (int i = 0; i < GRAPH_LAUNCH_ITERATIONS; i++)
    {
        hipGraphLaunch(clonedGraphExec, streamForGraph);
        hipStreamSynchronize(streamForGraph);
        verify_result(h_array, h_mask, h_result, n);
    }
    cout << "Done! Verifyied successfully" << endl;

    cout << "\nTime taken by using CUDA GRAPH in ms : " << milliseconds / GRAPH_LAUNCH_ITERATIONS << endl;
    
    hipGraphExecDestroy(graphExec);
    hipGraphExecDestroy(clonedGraphExec);
    hipGraphDestroy(graph);
    hipGraphDestroy(clonedGraph);
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
    hipStreamDestroy(streamForGraph);
    hipEventDestroy(forkStreamEvent);
    hipEventDestroy(memcpyEvent);
    hipEventDestroy(start);
    hipEventDestroy(stop);

}

void NormalConvolution(int* h_array, int* h_mask, int* h_result, int* d_array, int* d_result, const int n, int THREADS, int GRID, size_t bytes_n) {


    hipMemcpy(d_array, h_array, bytes_n, hipMemcpyHostToDevice);

    convolution_1d << <GRID, THREADS >> > (d_array, d_result, n);

    hipMemcpy(h_result, d_result, bytes_n, hipMemcpyDeviceToHost);

    verify_result(h_array, h_mask, h_result, n);

}

int main() {

    int n = 1 << 20;

    int bytes_n = n * sizeof(int);
    size_t bytes_m = MASK_LENGTH * sizeof(int);

    int* h_array = new int[n];
    int* h_mask = new int[MASK_LENGTH];
    int* h_result = new int[n];

    int* d_array, * d_result;
    hipMalloc(&d_array, bytes_n);
    hipMalloc(&d_result, bytes_n);
    
    initialize_vector(h_array, n);
    initialize_vector(h_mask, MASK_LENGTH);
    hipMemcpyToSymbol(HIP_SYMBOL(mask), h_mask, bytes_m);

    int THREADS = 256;

    int GRID = (n + THREADS - 1) / THREADS;

    cout << "Normal Convolution" << endl;
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    
    for (int i = 0; i < GRAPH_LAUNCH_ITERATIONS; i++) {
        NormalConvolution(h_array, h_mask, h_result, d_array, d_result, n, THREADS, GRID, bytes_n);
        hipStreamSynchronize(0);
    }
    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "\nTime taken without CUDA GRAPH in ms : " << milliseconds / GRAPH_LAUNCH_ITERATIONS << endl;
    
    cout << "----------------------------------------------------" << endl;
    
    cout << "Convolution using CUDA GRAPHS (Stream Capture)" << endl;
    
    StreamCaptureConvolution(h_array, h_mask, h_result, d_array, d_result, n, THREADS, GRID, bytes_n, bytes_m);

    cout << "----------------------------------------------------\n" << endl;
    cout << "\nThe time is the average time of all the kernel launchs. The total kernel launches are " << GRAPH_LAUNCH_ITERATIONS << endl;

    delete[] h_array;
    delete[] h_result;
    delete[] h_mask;
    hipFree(d_result);
    hipFree(d_array);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}