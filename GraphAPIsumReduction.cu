#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <vector>

#define SIZE 1024
#define SHMEM_SIZE 1024*sizeof(int)
#define GRAPH_LAUNCH_ITERATIONS  300

using std::cout;
using std::endl;

__global__ void sum_reduction(int* v, int* v_r) {
	__shared__ int partial_sum[SHMEM_SIZE];

	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	partial_sum[threadIdx.x] = v[tid];
	__syncthreads();

	for (int s = blockDim.x / 2; s > 0; s >>= 1) {
		if (threadIdx.x < s) {
			partial_sum[threadIdx.x] += partial_sum[threadIdx.x + s];
		}
		__syncthreads();
	}

	if (threadIdx.x == 0) {
		v_r[blockIdx.x] = partial_sum[0];
	}
}

void initialize_vector(int* v, int n) {
	for (int i = 0; i < n; i++) {
		v[i] = rand() % 100;
	}
}

void verify_result(int* a, int* r, const int N) {
	int sum = 0;
	for (int i = 0; i < N; i++) {
		sum += a[i];
	}
	if (sum != r[0]) {
		cout << "NOT SUCCESSFUL" << endl;
		exit(0);
	}
}

void cudaGraphAPIsumReduction(int* h_v, int* h_v_r, int* d_v, int* d_v_r, int TB_SIZE, int GRID_SIZE, int n) {

	hipStream_t streamForGraph;
	hipGraph_t graph;
	std::vector<hipGraphNode_t> nodeDependencies;
	hipGraphNode_t memcpyNode, kernelNode, memsetNode;
	double result_h = 0.0;

	hipStreamCreateWithFlags(&streamForGraph, hipStreamNonBlocking);

	hipKernelNodeParams kernelNodeParams = { 0 };
	hipMemcpy3DParms memcpyParams = { 0 };
	hipMemsetParams memsetParams = { 0 };

	//Adding memcpyParams node
	memcpyParams.srcArray = NULL;
	memcpyParams.srcPos = make_hipPos(0, 0, 0);
	memcpyParams.srcPtr = make_hipPitchedPtr(h_v, sizeof(int) * n, n, 1);
	memcpyParams.dstArray = NULL;
	memcpyParams.dstPos = make_hipPos(0, 0, 0);
	memcpyParams.dstPtr = make_hipPitchedPtr(d_v, sizeof(int) * n, n, 1);
	memcpyParams.extent = make_hipExtent(sizeof(int) * n, 1, 1);
	memcpyParams.kind = hipMemcpyHostToDevice;

	//Adding memsetParams node
	memsetParams.dst = (void*)d_v_r;
	memsetParams.value = 0;
	memsetParams.pitch = 0;
	memsetParams.elementSize = sizeof(int);
	memsetParams.width = n;
	memsetParams.height = 1;

	hipGraphCreate(&graph, 0);
	hipGraphAddMemcpyNode(&memcpyNode, graph, NULL, 0, &memcpyParams);
	hipGraphAddMemsetNode(&memsetNode, graph, NULL, 0, &memsetParams);
	nodeDependencies.push_back(memsetNode);
	nodeDependencies.push_back(memcpyNode);


	//Adding Kernal node
	void* kernelArgs[2] = { (void*)&d_v, (void*)&d_v_r };
	kernelNodeParams.func = (void*)sum_reduction;
	kernelNodeParams.gridDim = dim3(GRID_SIZE, 1, 1);
	kernelNodeParams.blockDim = dim3(TB_SIZE, 1, 1);
	kernelNodeParams.sharedMemBytes = 0;
	kernelNodeParams.kernelParams = (void**)kernelArgs;
	kernelNodeParams.extra = NULL;

	hipGraphAddKernelNode(&kernelNode, graph, nodeDependencies.data(), nodeDependencies.size(), &kernelNodeParams);
	nodeDependencies.clear();
	nodeDependencies.push_back(kernelNode);

	//Adding Kernal node
	memset(&kernelNodeParams, 0, sizeof(kernelNodeParams));
	void* kernelArgs2[2] = { (void*)&d_v_r, (void*)&d_v_r };
	kernelNodeParams.func = (void*)sum_reduction;
	kernelNodeParams.gridDim = dim3(1, 1, 1);
	kernelNodeParams.blockDim = dim3(TB_SIZE, 1, 1);
	kernelNodeParams.sharedMemBytes = 0;
	kernelNodeParams.kernelParams = (void**)kernelArgs2;
	kernelNodeParams.extra = NULL;

	hipGraphAddKernelNode(&kernelNode, graph, nodeDependencies.data(), nodeDependencies.size(), &kernelNodeParams);
	nodeDependencies.clear();
	nodeDependencies.push_back(kernelNode);

	//Adding memcpyParams node
	memset(&memcpyParams, 0, sizeof(memcpyParams));
	memcpyParams.srcArray = NULL;
	memcpyParams.srcPos = make_hipPos(0, 0, 0);
	memcpyParams.srcPtr = make_hipPitchedPtr(d_v_r, sizeof(int) * n, n, 1);
	memcpyParams.dstArray = NULL;
	memcpyParams.dstPos = make_hipPos(0, 0, 0);
	memcpyParams.dstPtr = make_hipPitchedPtr(h_v_r, sizeof(int) * n, n, 1);
	memcpyParams.extent = make_hipExtent(sizeof(int) * n, 1, 1);
	memcpyParams.kind = hipMemcpyDeviceToHost;

	hipGraphAddMemcpyNode(&memcpyNode, graph, nodeDependencies.data(), nodeDependencies.size(), &memcpyParams);
	nodeDependencies.clear();
	nodeDependencies.push_back(memcpyNode);

	hipGraphNode_t* nodes = NULL;
	size_t numNodes = 0;
	hipGraphGetNodes(graph, nodes, &numNodes);
	cout << "Num of nodes in the graph created manually = " << numNodes << endl;

	hipGraphExec_t graphExec;
	hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0);

	hipGraph_t clonedGraph;
	hipGraphExec_t clonedGraphExec;
	hipGraphClone(&clonedGraph, graph);
	hipGraphInstantiate(&clonedGraphExec, clonedGraph, NULL, NULL, 0);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	
	for (int i = 0; i < GRAPH_LAUNCH_ITERATIONS; i++)
	{
		hipGraphLaunch(graphExec, streamForGraph);
		hipStreamSynchronize(streamForGraph);
		verify_result(h_v, h_v_r, n);
	}
	
	hipDeviceSynchronize();
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	cout << "Verifying Cloned Graph ..." << endl;
	for (int i = 0; i < GRAPH_LAUNCH_ITERATIONS; i++)
	{
		hipGraphLaunch(clonedGraphExec, streamForGraph);
		hipStreamSynchronize(streamForGraph);
		verify_result(h_v, h_v_r, n);
	}
	cout << "Done! Verifyied successfully" << endl;

	cout << "\nTime taken by using CUDA GRAPH in ms : " << milliseconds / GRAPH_LAUNCH_ITERATIONS << endl;

	hipGraphExecDestroy(graphExec);
	hipGraphExecDestroy(clonedGraphExec);
	hipGraphDestroy(graph);
	hipGraphDestroy(clonedGraph);
	hipStreamDestroy(streamForGraph);
	hipEventDestroy(start);
	hipEventDestroy(stop);

}



void sumReduction(int* h_v, int* h_v_r, int* d_v, int* d_v_r, int TB_SIZE, int GRID_SIZE, int n) {

	hipMemcpy(d_v, h_v, n * sizeof(int), hipMemcpyHostToDevice);

	sum_reduction << <GRID_SIZE, TB_SIZE >> > (d_v, d_v_r);

	sum_reduction << <1, TB_SIZE >> > (d_v_r, d_v_r);

	hipMemcpy(h_v_r, d_v_r, n * sizeof(int), hipMemcpyDeviceToHost);

	verify_result(h_v, h_v_r, n);

}

int main() {

	int n = 1 << 20;
	size_t bytes = n * sizeof(int);

	int* h_v, * h_v_r;
	int* d_v, * d_v_r;

	h_v = (int*)malloc(bytes);
	h_v_r = (int*)malloc(bytes);
	hipMalloc(&d_v, bytes);
	hipMalloc(&d_v_r, bytes);

	initialize_vector(h_v, n);

	int	TB_SIZE = SIZE;
	int GRID_SIZE = (n + TB_SIZE - 1) / TB_SIZE;
	
	cout << "Normal Sum Reduction\n" << endl;
	
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	
	for (int i = 0; i < GRAPH_LAUNCH_ITERATIONS; i++) {
		sumReduction(h_v, h_v_r, d_v, d_v_r, TB_SIZE, GRID_SIZE, n);
		hipStreamSynchronize(0);
	}
	
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	cout << "Time taken without CUDA GRAPH in ms : " << milliseconds/ GRAPH_LAUNCH_ITERATIONS << endl;

	cout << "----------------------------------------------------\n" << endl;

	cout << "Sum Reduction using CUDA GRAPHS (Graph API)\n" << endl;

	cudaGraphAPIsumReduction(h_v, h_v_r, d_v, d_v_r, TB_SIZE, GRID_SIZE, n);
	
	cout << "----------------------------------------------------\n" << endl;
	cout << "\nThe time is the average time of all the kernel launchs. The total kernel launches are " << GRAPH_LAUNCH_ITERATIONS << endl;
	
	hipFree(d_v);
	hipFree(d_v_r);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	return 0;
}

