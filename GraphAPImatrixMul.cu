#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <>
#include <vector>
#define GRAPH_LAUNCH_ITERATIONS  300


using std::endl;
using std::cout;


void init_matrix(int* a, const int N) {
    for (int i = 0; i < N * N; i++) {
        a[i] = rand() % 100;
    }
}

__global__ void matrixMul(int* a, int* b, int* c, const int N) {

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    c[row * N + col] = 0;
    for (int k = 0; k < N; k++) {
        c[row * N + col] += a[row * N + k] * b[k * N + col];
    }
}

void verify_result(int* a, int* b, int* c, const int N) {

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            int tmp = 0;
            for (int k = 0; k < N; k++) {
                tmp += a[i * N + k] * b[k * N + j];
            }
            if (tmp != c[i * N + j]) {
                cout << "NOT SUCCESSFUL" << endl;
                exit(0);
            }
        }
    }
}

void cudaGraphAPImatrixMul(int* h_a, int* h_b, int* h_c, int* d_a, int* d_b, int* d_c, dim3 threads, dim3 blocks, int N) {

    hipStream_t streamForGraph;
    hipGraph_t graph;
    std::vector<hipGraphNode_t> nodeDependencies;
    hipGraphNode_t memcpyNode, kernelNode, memsetNode;
    double result_h = 0.0;

    hipStreamCreateWithFlags(&streamForGraph, hipStreamNonBlocking);

    hipKernelNodeParams kernelNodeParams = { 0 };
    hipMemcpy3DParms memcpyParams = { 0 };
    hipMemsetParams memsetParams = { 0 };

    //Adding memcpyParams node
    memcpyParams.srcArray = NULL;
    memcpyParams.srcPos = make_hipPos(0, 0, 0);
    memcpyParams.srcPtr = make_hipPitchedPtr(h_a, sizeof(int) * N * N, N * N, 1);
    memcpyParams.dstArray = NULL;
    memcpyParams.dstPos = make_hipPos(0, 0, 0);
    memcpyParams.dstPtr = make_hipPitchedPtr(d_a, sizeof(int) * N * N, N * N, 1);
    memcpyParams.extent = make_hipExtent(sizeof(int) * N * N, 1, 1);
    memcpyParams.kind = hipMemcpyHostToDevice;

    //Adding memsetParams node
    memsetParams.dst = (void*)d_c;
    memsetParams.value = 0;
    memsetParams.pitch = 0;
    memsetParams.elementSize = sizeof(int);
    memsetParams.width = N * N;
    memsetParams.height = 1;

    hipGraphCreate(&graph, 0);
    hipGraphAddMemcpyNode(&memcpyNode, graph, NULL, 0, &memcpyParams);
    hipGraphAddMemsetNode(&memsetNode, graph, NULL, 0, &memsetParams);
    nodeDependencies.push_back(memsetNode);
    nodeDependencies.push_back(memcpyNode);

    //Adding memcpyParams node
    memset(&memcpyParams, 0, sizeof(memcpyParams));
    memcpyParams.srcArray = NULL;
    memcpyParams.srcPos = make_hipPos(0, 0, 0);
    memcpyParams.srcPtr = make_hipPitchedPtr(h_b, sizeof(int) * N * N, N * N, 1);
    memcpyParams.dstArray = NULL;
    memcpyParams.dstPos = make_hipPos(0, 0, 0);
    memcpyParams.dstPtr = make_hipPitchedPtr(d_b, sizeof(int) * N * N, N * N, 1);
    memcpyParams.extent = make_hipExtent(sizeof(int) * N * N, 1, 1);
    memcpyParams.kind = hipMemcpyHostToDevice;

    hipGraphAddMemcpyNode(&memcpyNode, graph, NULL, 0, &memcpyParams);
    nodeDependencies.push_back(memcpyNode);

    //Adding Kernal node
    void* kernelArgs[4] = { (void*)&d_a, (void*)&d_b, &d_c, &N };
    kernelNodeParams.func = (void*)matrixMul;
    kernelNodeParams.gridDim = blocks;
    kernelNodeParams.blockDim = threads;
    kernelNodeParams.sharedMemBytes = 0;
    kernelNodeParams.kernelParams = (void**)kernelArgs;
    kernelNodeParams.extra = NULL;

    hipGraphAddKernelNode(&kernelNode, graph, nodeDependencies.data(), nodeDependencies.size(), &kernelNodeParams);
    nodeDependencies.clear();
    nodeDependencies.push_back(kernelNode);

    //Adding memcpyParams node
    memset(&memcpyParams, 0, sizeof(memcpyParams));
    memcpyParams.srcArray = NULL;
    memcpyParams.srcPos = make_hipPos(0, 0, 0);
    memcpyParams.srcPtr = make_hipPitchedPtr(d_c, sizeof(int) * N * N, N * N, 1);
    memcpyParams.dstArray = NULL;
    memcpyParams.dstPos = make_hipPos(0, 0, 0);
    memcpyParams.dstPtr = make_hipPitchedPtr(h_c, sizeof(int) * N * N, N * N, 1);
    memcpyParams.extent = make_hipExtent(sizeof(int) * N * N, 1, 1);
    memcpyParams.kind = hipMemcpyDeviceToHost;

    hipGraphAddMemcpyNode(&memcpyNode, graph, nodeDependencies.data(), nodeDependencies.size(), &memcpyParams);
    nodeDependencies.clear();
    nodeDependencies.push_back(memcpyNode);

    hipGraphNode_t* nodes = NULL;
    size_t numNodes = 0;
    hipGraphGetNodes(graph, nodes, &numNodes);
    cout << "\nNum of nodes in the graph created manually = " << numNodes << endl;

    hipGraphExec_t graphExec;
    hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0);

    hipGraph_t clonedGraph;
    hipGraphExec_t clonedGraphExec;
    hipGraphClone(&clonedGraph, graph);
    hipGraphInstantiate(&clonedGraphExec, clonedGraph, NULL, NULL, 0);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    for (int i = 0; i < GRAPH_LAUNCH_ITERATIONS; i++)
    {
        hipGraphLaunch(graphExec, streamForGraph);
        hipStreamSynchronize(streamForGraph);
        verify_result(h_a, h_b, h_c, N);
    }

    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    cout << "Verifying Cloned Graph ..." << endl;
    for (int i = 0; i < GRAPH_LAUNCH_ITERATIONS; i++)
    {
        hipGraphLaunch(clonedGraphExec, streamForGraph);
        hipStreamSynchronize(streamForGraph);
        verify_result(h_a, h_b, h_c, N);
    }
    cout << "Done! Verifyied successfully" << endl;

    cout << "\nTime taken by using CUDA GRAPH in ms : " << milliseconds / GRAPH_LAUNCH_ITERATIONS << endl;

    hipGraphExecDestroy(graphExec);
    hipGraphExecDestroy(clonedGraphExec);
    hipGraphDestroy(graph);
    hipGraphDestroy(clonedGraph);
    hipStreamDestroy(streamForGraph);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}



void NormalMatrixMul(int* h_a, int* h_b, int* h_c, int* d_a, int* d_b, int* d_c, dim3 threads, dim3 blocks, int N) {

    hipMemcpy(d_a, h_a, N * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * N * sizeof(int), hipMemcpyHostToDevice);

    matrixMul << <blocks, threads >> > (d_a, d_b, d_c, N);

    hipMemcpy(h_c, d_c, N * N * sizeof(int), hipMemcpyDeviceToHost);

    verify_result(h_a, h_b, h_c, N);

}


int main() {

    int N = 1 << 6;

    size_t bytes = N * N * sizeof(int);

    int* h_a, * h_b, * h_c;

    h_a = new int[N * N];
    h_b = new int[N * N];
    h_c = new int[N * N];

    init_matrix(h_a, N);
    init_matrix(h_b, N);

    int* d_a, * d_b, * d_c;
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    int THREADS = 32;
    int BLOCKS = (N + THREADS - 1) / THREADS;

    dim3 threads(THREADS, THREADS);
    dim3 blocks(BLOCKS, BLOCKS);

    cout << "Normal Matrix Mul" << endl;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    for (int i = 0; i < GRAPH_LAUNCH_ITERATIONS; i++) {
        NormalMatrixMul(h_a, h_b, h_c, d_a, d_b, d_c, threads, blocks, N);
    }
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    cout << "\nTime taken without CUDA GRAPH in ms : " << milliseconds / GRAPH_LAUNCH_ITERATIONS << endl;
    
    cout << "----------------------------------------------------" << endl;
    
    cout << "Matrix Mul using CUDA GRAPHS (Graph API)" << endl;
    
    cudaGraphAPImatrixMul(h_a, h_b, h_c, d_a, d_b, d_c, threads, blocks, N);

    cout << "----------------------------------------------------\n" << endl;
    cout << "\nThe time is the average time of all the kernel launchs. The total kernel launches are " << GRAPH_LAUNCH_ITERATIONS << endl;

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}