#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime_api.h>
#include <iostream>

#define SIZE 1024
#define SHMEM_SIZE 1024*sizeof(int)
#define GRAPH_LAUNCH_ITERATIONS  3

using std::cout;
using std::endl;

__global__ void sum_reduction(int* v, int* v_r) {
	__shared__ int partial_sum[SHMEM_SIZE];

	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	partial_sum[threadIdx.x] = v[tid];
	__syncthreads();

	for (int s = blockDim.x / 2; s > 0; s >>= 1) {
		if (threadIdx.x < s) {
			partial_sum[threadIdx.x] += partial_sum[threadIdx.x + s];
		}
		__syncthreads();
	}

	if (threadIdx.x == 0) {
		v_r[blockIdx.x] = partial_sum[0];
	}
}

void initialize_vector(int* v, int n) {
	for (int i = 0; i < n; i++) {
		v[i] = rand() % 100;
	}
}

void verify_result(int* a, int* r, const int N) {
	int sum = 0;
	for (int i = 0; i < N; i++) {
		sum += a[i];
	}
	if (sum != r[0]) {
		cout << "NOT SUCCESSFUL" << endl;
		exit(0);
	}
}

void cudaGraphsumReduction(int* h_v, int* h_v_r, int* d_v, int* d_v_r, int TB_SIZE, int GRID_SIZE, int n) {

	hipStream_t stream1, stream2, streamForGraph;
	hipEvent_t forkStreamEvent, memsetEvent;
	hipGraph_t graph;
	int result = 0;

	checkCudaErrors(hipStreamCreateWithFlags(&stream1, hipStreamNonBlocking));
	checkCudaErrors(hipStreamCreateWithFlags(&stream2, hipStreamNonBlocking));
	checkCudaErrors(hipStreamCreateWithFlags(&streamForGraph, hipStreamNonBlocking));

	checkCudaErrors(hipEventCreate(&forkStreamEvent));
	checkCudaErrors(hipEventCreate(&memsetEvent));

	checkCudaErrors(hipStreamBeginCapture(stream1, hipStreamCaptureModeGlobal));
	checkCudaErrors(hipEventRecord(forkStreamEvent, stream1));
	checkCudaErrors(hipStreamWaitEvent(stream2, forkStreamEvent));

	checkCudaErrors(hipMemcpyAsync(d_v, h_v, n * sizeof(int), hipMemcpyDefault, stream1));

	checkCudaErrors(hipMemsetAsync(d_v_r, 0, n * sizeof(int), stream2));
	checkCudaErrors(hipEventRecord(memsetEvent, stream2));
	checkCudaErrors(hipStreamWaitEvent(stream1, memsetEvent));

	sum_reduction << <GRID_SIZE, TB_SIZE, 0, stream1 >> > (d_v, d_v_r);

	sum_reduction << <1, TB_SIZE, 0, stream1 >> > (d_v_r, d_v_r);

	checkCudaErrors(hipMemcpyAsync(h_v_r, d_v_r, n * sizeof(int), hipMemcpyDefault, stream1));

	checkCudaErrors(hipStreamEndCapture(stream1, &graph));

	hipGraphNode_t* nodes = NULL;
	size_t numNodes = 0;
	checkCudaErrors(hipGraphGetNodes(graph, nodes, &numNodes));
	cout << "\nNum of nodes in the graph created using stream capture API = " << numNodes << endl;

	hipGraphExec_t graphExec;
	checkCudaErrors(hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0));

	hipGraph_t clonedGraph;
	hipGraphExec_t clonedGraphExec;
	checkCudaErrors(hipGraphClone(&clonedGraph, graph));
	checkCudaErrors(hipGraphInstantiate(&clonedGraphExec, clonedGraph, NULL, NULL, 0));

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

	for (int i = 0; i < GRAPH_LAUNCH_ITERATIONS; i++)
	{
		hipGraphLaunch(graphExec, streamForGraph);
		hipStreamSynchronize(streamForGraph);
		verify_result(h_v, h_v_r, n);
	}

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	cout << "\nVerifying Cloned Graph ... " << endl;
	for (int i = 0; i < GRAPH_LAUNCH_ITERATIONS; i++)
	{
		checkCudaErrors(hipGraphLaunch(clonedGraphExec, streamForGraph));
		checkCudaErrors(hipStreamSynchronize(streamForGraph));
		verify_result(h_v, h_v_r, n);
	}
	cout << "Done! Verifyied successfully" << endl;

	cout << "\nTime taken by using CUDA GRAPH in ms : " << milliseconds / GRAPH_LAUNCH_ITERATIONS << endl;

	checkCudaErrors(hipStreamSynchronize(streamForGraph));

	checkCudaErrors(hipGraphExecDestroy(graphExec));
	checkCudaErrors(hipGraphExecDestroy(clonedGraphExec));
	checkCudaErrors(hipGraphDestroy(graph));
	checkCudaErrors(hipGraphDestroy(clonedGraph));
	checkCudaErrors(hipStreamDestroy(stream1));
	checkCudaErrors(hipStreamDestroy(stream2));
	checkCudaErrors(hipStreamDestroy(streamForGraph));
	checkCudaErrors(hipEventDestroy(start));
	checkCudaErrors(hipEventDestroy(stop));
}



void sumReduction(int* h_v, int* h_v_r, int* d_v, int* d_v_r, int TB_SIZE, int GRID_SIZE, int n) {

	hipMemcpy(d_v, h_v, n * sizeof(int), hipMemcpyHostToDevice);

	sum_reduction << <GRID_SIZE, TB_SIZE >> > (d_v, d_v_r);

	sum_reduction << <1, TB_SIZE >> > (d_v_r, d_v_r);

	hipMemcpy(h_v_r, d_v_r, n * sizeof(int), hipMemcpyDeviceToHost);

	verify_result(h_v, h_v_r, n);

}

int main() {

	int n = 1 << 20;
	size_t bytes = n * sizeof(int);

	int* h_v, * h_v_r;
	int* d_v, * d_v_r;

	h_v = (int*)malloc(bytes);
	h_v_r = (int*)malloc(bytes);
	checkCudaErrors(hipMalloc(&d_v, bytes));
	checkCudaErrors(hipMalloc(&d_v_r, bytes));

	initialize_vector(h_v, n);
	int	TB_SIZE = SIZE;
	int GRID_SIZE = (n + TB_SIZE - 1) / TB_SIZE;

	cout << "Normal Sum Reduction" << endl;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

	for (int i = 0; i < GRAPH_LAUNCH_ITERATIONS; i++) {
		sumReduction(h_v, h_v_r, d_v, d_v_r, TB_SIZE, GRID_SIZE, n);
		hipStreamSynchronize(0);
	}

	hipDeviceSynchronize();
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	cout << "\nTime taken without CUDA GRAPH in ms : " << milliseconds / GRAPH_LAUNCH_ITERATIONS << endl;

	cout << "----------------------------------------------------" << endl;

	cout << "Sum Reduction using CUDA GRAPHS (Stream Capture)" << endl;
	cudaGraphsumReduction(h_v, h_v_r, d_v, d_v_r, TB_SIZE, GRID_SIZE, n);

	cout << "----------------------------------------------------\n" << endl;
	cout << "\nThe time is the average time of all the kernel launchs. The total kernel launches are " << GRAPH_LAUNCH_ITERATIONS << endl;

	checkCudaErrors(hipFree(d_v));
	checkCudaErrors(hipFree(d_v_r));
	return 0;
}

