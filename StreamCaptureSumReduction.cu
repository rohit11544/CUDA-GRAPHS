#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <iostream>

#define SIZE 1024
#define SHMEM_SIZE 1024*sizeof(int)
#define GRAPH_LAUNCH_ITERATIONS  3

using std::cout;
using std::endl;

__global__ void sum_reduction(int* v, int* v_r) {
	__shared__ int partial_sum[SHMEM_SIZE];

	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	partial_sum[threadIdx.x] = v[tid];
	__syncthreads();

	for (int s = blockDim.x / 2; s > 0; s >>= 1) {
		if (threadIdx.x < s) {
			partial_sum[threadIdx.x] += partial_sum[threadIdx.x + s];
		}
		__syncthreads();
	}

	if (threadIdx.x == 0) {
		v_r[blockIdx.x] = partial_sum[0];
	}
}

void initialize_vector(int* v, int n) {
	for (int i = 0; i < n; i++) {
		v[i] = rand() % 100;
	}
}

void verify_result(int* a, int* r, const int N) {
	int sum = 0;
	for (int i = 0; i < N; i++) {
		sum += a[i];
	}
	if (sum != r[0]) {
		cout << "NOT SUCCESSFUL" << endl;
		exit(0);
	}
}

void cudaGraphsumReduction(int* h_v, int* h_v_r, int* d_v, int* d_v_r, int TB_SIZE, int GRID_SIZE, int n) {

	hipStream_t stream1, stream2, streamForGraph;
	hipEvent_t forkStreamEvent, memsetEvent;
	hipGraph_t graph;
	int result = 0;

	hipStreamCreateWithFlags(&stream1, hipStreamNonBlocking);
	hipStreamCreateWithFlags(&stream2, hipStreamNonBlocking);
	hipStreamCreateWithFlags(&streamForGraph, hipStreamNonBlocking);

	hipEventCreate(&forkStreamEvent);
	hipEventCreate(&memsetEvent);

	hipStreamBeginCapture(stream1, hipStreamCaptureModeGlobal);
	hipEventRecord(forkStreamEvent, stream1);
	hipStreamWaitEvent(stream2, forkStreamEvent);

	hipMemcpyAsync(d_v, h_v, n * sizeof(int), hipMemcpyDefault, stream1);

	hipMemsetAsync(d_v_r, 0, n * sizeof(int), stream2);
	hipEventRecord(memsetEvent, stream2);
	hipStreamWaitEvent(stream1, memsetEvent);

	sum_reduction << <GRID_SIZE, TB_SIZE, 0, stream1 >> > (d_v, d_v_r);

	sum_reduction << <1, TB_SIZE, 0, stream1 >> > (d_v_r, d_v_r);

	hipMemcpyAsync(h_v_r, d_v_r, n * sizeof(int), hipMemcpyDefault, stream1);

	hipStreamEndCapture(stream1, &graph);

	hipGraphNode_t* nodes = NULL;
	size_t numNodes = 0;
	hipGraphGetNodes(graph, nodes, &numNodes);
	cout << "\nNum of nodes in the graph created using stream capture API = " << numNodes << endl;

	hipGraphExec_t graphExec;
	hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0);

	hipGraph_t clonedGraph;
	hipGraphExec_t clonedGraphExec;
	hipGraphClone(&clonedGraph, graph);
	hipGraphInstantiate(&clonedGraphExec, clonedGraph, NULL, NULL, 0);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

	for (int i = 0; i < GRAPH_LAUNCH_ITERATIONS; i++)
	{
		hipGraphLaunch(graphExec, streamForGraph);
		hipStreamSynchronize(streamForGraph);
		verify_result(h_v, h_v_r, n);
	}
	
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	
	cout << "\nVerifying Cloned Graph ... " << endl;
	for (int i = 0; i < GRAPH_LAUNCH_ITERATIONS; i++)
	{
		hipGraphLaunch(clonedGraphExec, streamForGraph);
		hipStreamSynchronize(streamForGraph);
		verify_result(h_v, h_v_r, n);
	}
	cout << "Done! Verifyied successfully" << endl;

	cout << "\nTime taken by using CUDA GRAPH in ms : " << milliseconds / GRAPH_LAUNCH_ITERATIONS << endl;

	hipStreamSynchronize(streamForGraph);

	hipGraphExecDestroy(graphExec);
	hipGraphExecDestroy(clonedGraphExec);
	hipGraphDestroy(graph);
	hipGraphDestroy(clonedGraph);
	hipStreamDestroy(stream1);
	hipStreamDestroy(stream2);
	hipStreamDestroy(streamForGraph);
	hipEventDestroy(start);
	hipEventDestroy(stop);
}



void sumReduction(int* h_v, int* h_v_r, int* d_v, int* d_v_r, int TB_SIZE, int GRID_SIZE, int n) {

	hipMemcpy(d_v, h_v, n * sizeof(int), hipMemcpyHostToDevice);

	sum_reduction << <GRID_SIZE, TB_SIZE >> > (d_v, d_v_r);

	sum_reduction << <1, TB_SIZE >> > (d_v_r, d_v_r);

	hipMemcpy(h_v_r, d_v_r, n * sizeof(int), hipMemcpyDeviceToHost);

	verify_result(h_v, h_v_r, n);

}

int main() {

	int n = 1 << 20;
	size_t bytes = n * sizeof(int);

	int* h_v, * h_v_r;
	int* d_v, * d_v_r;

	h_v = (int*)malloc(bytes);
	h_v_r = (int*)malloc(bytes);
	hipMalloc(&d_v, bytes);
	hipMalloc(&d_v_r, bytes);

	initialize_vector(h_v, n);
	int	TB_SIZE = SIZE;
	int GRID_SIZE = (n + TB_SIZE - 1) / TB_SIZE;

	cout << "Normal Sum Reduction" << endl;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

	for (int i = 0; i < GRAPH_LAUNCH_ITERATIONS; i++) {
		sumReduction(h_v, h_v_r, d_v, d_v_r, TB_SIZE, GRID_SIZE, n);
		hipStreamSynchronize(0);
	}

	hipDeviceSynchronize();
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	cout << "\nTime taken without CUDA GRAPH in ms : " << milliseconds / GRAPH_LAUNCH_ITERATIONS << endl;

	cout << "----------------------------------------------------" << endl;
	
	cout << "Sum Reduction using CUDA GRAPHS (Stream Capture)" << endl;
	cudaGraphsumReduction(h_v, h_v_r, d_v, d_v_r, TB_SIZE, GRID_SIZE, n);

	cout << "----------------------------------------------------\n" << endl;
	cout << "\nThe time is the average time of all the kernel launchs. The total kernel launches are " << GRAPH_LAUNCH_ITERATIONS << endl;

	hipFree(d_v);
	hipFree(d_v_r);
	return 0;
}

