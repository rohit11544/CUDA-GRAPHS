#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#define GRAPH_LAUNCH_ITERATIONS  3


using std::endl;
using std::cout;


void init_matrix(int* a, const int N) {
    for (int i = 0; i < N * N; i++) {
        a[i] = rand() % 100;
    }
}

__global__ void matrixMul(int* a, int* b, int* c, const int N) {

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    c[row * N + col] = 0;
    for (int k = 0; k < N; k++) {
        c[row * N + col] += a[row * N + k] * b[k * N + col];
    }
}

void verify_result(int* a, int* b, int* c, const int N) {

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            int tmp = 0;
            for (int k = 0; k < N; k++) {
                tmp += a[i * N + k] * b[k * N + j];
            }
            if (tmp != c[i * N + j]) {
                cout << "NOT SUCCESSFUL" << endl;
                exit(0);
            }
        }
    }
}



void StreamCaptureMatrixMul(int* h_a, int* h_b, int* h_c, int* d_a, int* d_b, int* d_c, dim3 threads, dim3 blocks, int N) {

    hipStream_t stream1, stream2, stream3, streamForGraph;
    hipEvent_t forkStreamEvent, memcpyEvent, memsetEvent;
    hipGraph_t graph;
    int result = 0;

    checkCudaErrors(hipStreamCreateWithFlags(&stream1, hipStreamNonBlocking));
    checkCudaErrors(hipStreamCreateWithFlags(&stream2, hipStreamNonBlocking));
    checkCudaErrors(hipStreamCreateWithFlags(&stream3, hipStreamNonBlocking));
    checkCudaErrors(hipStreamCreateWithFlags(&streamForGraph, hipStreamNonBlocking));

    checkCudaErrors(hipEventCreate(&forkStreamEvent));
    checkCudaErrors(hipEventCreate(&memcpyEvent));
    checkCudaErrors(hipEventCreate(&memsetEvent));

    checkCudaErrors(hipStreamBeginCapture(stream1, hipStreamCaptureModeGlobal));
    checkCudaErrors(hipEventRecord(forkStreamEvent, stream1));
    checkCudaErrors(hipStreamWaitEvent(stream2, forkStreamEvent));
    checkCudaErrors(hipStreamWaitEvent(stream3, forkStreamEvent));

    checkCudaErrors(hipMemcpyAsync(d_a, h_a, N * N * sizeof(int), hipMemcpyDefault, stream1));
    checkCudaErrors(hipMemcpyAsync(d_b, h_b, N * N * sizeof(int), hipMemcpyDefault, stream2));
    checkCudaErrors(hipEventRecord(memcpyEvent, stream2));
    checkCudaErrors(hipStreamWaitEvent(stream1, memcpyEvent));

    checkCudaErrors(hipMemsetAsync(d_c, 0, N * N * sizeof(int), stream3));
    checkCudaErrors(hipEventRecord(memsetEvent, stream3));
    checkCudaErrors(hipStreamWaitEvent(stream1, memsetEvent));

    matrixMul << <blocks, threads, 0, stream1 >> > (d_a, d_b, d_c, N);

    checkCudaErrors(hipMemcpyAsync(h_c, d_c, N * N * sizeof(int), hipMemcpyDefault, stream1));

    checkCudaErrors(hipStreamEndCapture(stream1, &graph));

    hipGraphNode_t* nodes = NULL;
    size_t numNodes = 0;
    checkCudaErrors(hipGraphGetNodes(graph, nodes, &numNodes));
    cout << "Num of nodes in the graph created using stream capture API = " << numNodes << endl;

    hipGraphExec_t graphExec;
    checkCudaErrors(hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0));

    hipGraph_t clonedGraph;
    hipGraphExec_t clonedGraphExec;
    checkCudaErrors(hipGraphClone(&clonedGraph, graph));
    checkCudaErrors(hipGraphInstantiate(&clonedGraphExec, clonedGraph, NULL, NULL, 0));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    for (int i = 0; i < GRAPH_LAUNCH_ITERATIONS; i++)
    {
        hipGraphLaunch(graphExec, streamForGraph);
        hipStreamSynchronize(streamForGraph);
        verify_result(h_a, h_b, h_c, N);
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    cout << "\nVerifying Cloned Graph ... " << endl;
    for (int i = 0; i < GRAPH_LAUNCH_ITERATIONS; i++)
    {
        checkCudaErrors(hipGraphLaunch(clonedGraphExec, streamForGraph));
        checkCudaErrors(hipStreamSynchronize(streamForGraph));
        verify_result(h_a, h_b, h_c, N);
    }
    cout << "Done! Verifyied successfully" << endl;

    cout << "\nTime taken by using CUDA GRAPH in ms : " << milliseconds / GRAPH_LAUNCH_ITERATIONS << endl;

    checkCudaErrors(hipStreamSynchronize(streamForGraph));
    checkCudaErrors(hipGraphExecDestroy(graphExec));
    checkCudaErrors(hipGraphExecDestroy(clonedGraphExec));
    checkCudaErrors(hipGraphDestroy(graph));
    checkCudaErrors(hipGraphDestroy(clonedGraph));
    checkCudaErrors(hipStreamDestroy(stream1));
    checkCudaErrors(hipStreamDestroy(stream2));
    checkCudaErrors(hipStreamDestroy(streamForGraph));
    checkCudaErrors(hipEventDestroy(memcpyEvent));
    checkCudaErrors(hipEventDestroy(memsetEvent));
    checkCudaErrors(hipEventDestroy(forkStreamEvent));
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));

}



void NormalMatrixMul(int* h_a, int* h_b, int* h_c, int* d_a, int* d_b, int* d_c, dim3 threads, dim3 blocks, int N) {

    hipMemcpy(d_a, h_a, N * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * N * sizeof(int), hipMemcpyHostToDevice);

    matrixMul << <blocks, threads >> > (d_a, d_b, d_c, N);

    hipMemcpy(h_c, d_c, N * N * sizeof(int), hipMemcpyDeviceToHost);

    verify_result(h_a, h_b, h_c, N);

}


int main() {

    int N = 1 << 10;

    size_t bytes = N * N * sizeof(int);

    int* h_a, * h_b, * h_c;

    h_a = new int[N * N];
    h_b = new int[N * N];
    h_c = new int[N * N];

    int* d_a, * d_b, * d_c;
    checkCudaErrors(hipMalloc(&d_a, bytes));
    checkCudaErrors(hipMalloc(&d_b, bytes));
    checkCudaErrors(hipMalloc(&d_c, bytes));

    init_matrix(h_a, N);
    init_matrix(h_b, N);

    int THREADS = 32;
    int BLOCKS = (N + THREADS - 1) / THREADS;

    dim3 threads(THREADS, THREADS);
    dim3 blocks(BLOCKS, BLOCKS);

    cout << "Normal Matrix Mul" << endl;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    for (int i = 0; i < GRAPH_LAUNCH_ITERATIONS; i++) {
        NormalMatrixMul(h_a, h_b, h_c, d_a, d_b, d_c, threads, blocks, N);
    }

    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "\nTime taken without CUDA GRAPH in ms : " << milliseconds / GRAPH_LAUNCH_ITERATIONS << endl;

    cout << "----------------------------------------------------" << endl;

    cout << "Matrix Mul using CUDA GRAPHS (Stream Capture)" << endl;

    StreamCaptureMatrixMul(h_a, h_b, h_c, d_a, d_b, d_c, threads, blocks, N);

    cout << "----------------------------------------------------\n" << endl;
    cout << "\nThe time is the average time of all the kernel launchs. The total kernel launches are " << GRAPH_LAUNCH_ITERATIONS << endl;

    checkCudaErrors(hipFree(d_a));
    checkCudaErrors(hipFree(d_b));
    checkCudaErrors(hipFree(d_c));
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));

    return 0;
}
