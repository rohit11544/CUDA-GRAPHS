#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <>
#define GRAPH_LAUNCH_ITERATIONS  3


using std::endl;
using std::cout;


void init_matrix(int* a, const int N) {
    for (int i = 0; i < N * N; i++) {
        a[i] = rand() % 100;
    }
}

__global__ void matrixMul(int* a, int* b, int* c, const int N) {

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    c[row * N + col] = 0;
    for (int k = 0; k < N; k++) {
        c[row * N + col] += a[row * N + k] * b[k * N + col];
    }
}

void verify_result(int* a, int* b, int* c, const int N) {

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            int tmp = 0;
            for (int k = 0; k < N; k++) {
                tmp += a[i * N + k] * b[k * N + j];
            }
            if (tmp != c[i * N + j]) {
                cout << "NOT SUCCESSFUL" << endl;
                exit(0);
            }
        }
    }
}



void StreamCaptureMatrixMul(int* h_a, int* h_b, int* h_c, int* d_a, int* d_b, int* d_c, dim3 threads, dim3 blocks, int N) {

    hipStream_t stream1, stream2, stream3, streamForGraph;
    hipEvent_t forkStreamEvent, memcpyEvent, memsetEvent;
    hipGraph_t graph;
    int result = 0;

    hipStreamCreateWithFlags(&stream1, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&stream2, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&stream3, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&streamForGraph, hipStreamNonBlocking);

    hipEventCreate(&forkStreamEvent);
    hipEventCreate(&memcpyEvent);
    hipEventCreate(&memsetEvent);

    hipStreamBeginCapture(stream1, hipStreamCaptureModeGlobal);
    hipEventRecord(forkStreamEvent, stream1);
    hipStreamWaitEvent(stream2, forkStreamEvent);
    hipStreamWaitEvent(stream3, forkStreamEvent);

    hipMemcpyAsync(d_a, h_a, N * N * sizeof(int), hipMemcpyDefault, stream1);
    hipMemcpyAsync(d_b, h_b, N * N * sizeof(int), hipMemcpyDefault, stream2);
    hipEventRecord(memcpyEvent, stream2);
    hipStreamWaitEvent(stream1, memcpyEvent);

    hipMemsetAsync(d_c, 0, N * N * sizeof(int), stream3);
    hipEventRecord(memsetEvent, stream3);
    hipStreamWaitEvent(stream1, memsetEvent);

    matrixMul << <blocks, threads, 0, stream1 >> > (d_a, d_b, d_c, N);

    hipMemcpyAsync(h_c, d_c, N * N * sizeof(int), hipMemcpyDefault, stream1);

    hipStreamEndCapture(stream1, &graph);

    hipGraphNode_t* nodes = NULL;
    size_t numNodes = 0;
    hipGraphGetNodes(graph, nodes, &numNodes);
    cout << "Num of nodes in the graph created using stream capture API = " << numNodes << endl;

    hipGraphExec_t graphExec;
    hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0);

    hipGraph_t clonedGraph;
    hipGraphExec_t clonedGraphExec;
    hipGraphClone(&clonedGraph, graph);
    hipGraphInstantiate(&clonedGraphExec, clonedGraph, NULL, NULL, 0);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    
    for (int i = 0; i < GRAPH_LAUNCH_ITERATIONS; i++)
    {
        hipGraphLaunch(graphExec, streamForGraph);
        hipStreamSynchronize(streamForGraph);
        verify_result(h_a, h_b, h_c, N);
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    cout << "\nVerifying Cloned Graph ... " << endl;
    for (int i = 0; i < GRAPH_LAUNCH_ITERATIONS; i++)
    {
        hipGraphLaunch(clonedGraphExec, streamForGraph);
        hipStreamSynchronize(streamForGraph);
        verify_result(h_a, h_b, h_c, N);
    }
    cout << "Done! Verifyied successfully" << endl;

    cout << "\nTime taken by using CUDA GRAPH in ms : " << milliseconds / GRAPH_LAUNCH_ITERATIONS << endl;

    hipStreamSynchronize(streamForGraph);
    hipGraphExecDestroy(graphExec);
    hipGraphExecDestroy(clonedGraphExec);
    hipGraphDestroy(graph);
    hipGraphDestroy(clonedGraph);
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
    hipStreamDestroy(streamForGraph);
    hipEventDestroy(memcpyEvent);
    hipEventDestroy(memsetEvent);
    hipEventDestroy(forkStreamEvent);
    hipEventDestroy(start);
    hipEventDestroy(stop);

}



void NormalMatrixMul(int* h_a, int* h_b, int* h_c, int* d_a, int* d_b, int* d_c, dim3 threads, dim3 blocks, int N) {

    hipMemcpy(d_a, h_a, N * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * N * sizeof(int), hipMemcpyHostToDevice);

    matrixMul << <blocks, threads >> > (d_a, d_b, d_c, N);

    hipMemcpy(h_c, d_c, N * N * sizeof(int), hipMemcpyDeviceToHost);

    verify_result(h_a, h_b, h_c, N);

}


int main() {

    int N = 1 << 10;

    size_t bytes = N * N * sizeof(int);

    int* h_a, * h_b, * h_c;

    h_a = new int[N * N];
    h_b = new int[N * N];
    h_c = new int[N * N];

    int* d_a, * d_b, * d_c;
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    init_matrix(h_a, N);
    init_matrix(h_b, N);

    int THREADS = 32;
    int BLOCKS = (N + THREADS - 1) / THREADS;

    dim3 threads(THREADS, THREADS);
    dim3 blocks(BLOCKS, BLOCKS);

    cout << "Normal Matrix Mul" << endl;
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    for (int i = 0; i < GRAPH_LAUNCH_ITERATIONS; i++) {
        NormalMatrixMul(h_a, h_b, h_c, d_a, d_b, d_c, threads, blocks, N);
    }
    
    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "\nTime taken without CUDA GRAPH in ms : " << milliseconds / GRAPH_LAUNCH_ITERATIONS << endl;

    cout << "----------------------------------------------------" << endl;
    
    cout << "Matrix Mul using CUDA GRAPHS (Stream Capture)" << endl;
    
    StreamCaptureMatrixMul(h_a, h_b, h_c, d_a, d_b, d_c, threads, blocks, N);

    cout << "----------------------------------------------------\n" << endl;
    cout << "\nThe time is the average time of all the kernel launchs. The total kernel launches are " << GRAPH_LAUNCH_ITERATIONS << endl;

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}